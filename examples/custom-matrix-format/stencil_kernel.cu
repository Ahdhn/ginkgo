#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2021, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include <cstdlib>

#include <ginkgo/ginkgo.hpp>


namespace {

template <typename T>
__device__ __forceinline__ T pitch(const T i, const T j, const T k, const T dim_x, const T dim_y, const T dim_z)
{    
    return k * dim_y * dim_z + j * dim_y + i;
}

// a parallel CUDA kernel that computes the application of a 3 point stencil
template <typename ValueType, typename BoundaryType>
__global__ void stencil_kernel_impl(std::size_t size, const BoundaryType *bd,
                                    const ValueType *b, ValueType *x)
{
    const auto thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id >= size) {
        return;
    }
    /*auto result = coefs[1] * b[thread_id];
    if (thread_id > 0) {
        result += coefs[0] * b[thread_id - 1];
    }
    if (thread_id < size - 1) {
        result += coefs[2] * b[thread_id + 1];
    }
    x[thread_id] = result;*/
}


}  // namespace


template <typename ValueType, typename BoundaryType>
void stencil_kernel(std::size_t size, const BoundaryType *bd, const ValueType *b, ValueType *x)
{
    constexpr auto block_size = 512;
    const auto grid_size = (size + block_size - 1) / block_size;
    stencil_kernel_impl<ValueType, BoundaryType><<<grid_size, block_size>>>(size, bd, b, x);
}

template void stencil_kernel<float, float>(std::size_t size, const float *bd, const float *b, float *x);
template void stencil_kernel<double, float>(std::size_t size, const float *bd, const double *b, double *x);
