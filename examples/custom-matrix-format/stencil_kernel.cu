#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2021, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include <cstdlib>

#include <ginkgo/ginkgo.hpp>


namespace {

template <typename T>
__device__ __forceinline__ T pitch(const T i, const T j, const T k, const T dim_x, const T dim_y, const T dim_z)
{    
    return k * dim_y * dim_z + j * dim_y + i;
}

// a parallel CUDA kernel that computes the application of a 3 point stencil
template <typename ValueType, typename BoundaryType>
__global__ void stencil_kernel_impl(std::size_t size, const BoundaryType *bd,
                                    const ValueType *b, ValueType *x, std::size_t dimx, std::size_t dimy, std::size_t dimz)
{
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id >= size) {
        return;
    }
    int k = thread_id / (dimy * dimx); 
    const int blah = thread_id - (k * dimx * dimy);
    int j = blah / dimx;
    int i = blah % dimx;
    
    //assert(pitch(i,j,k,dimx, dimy, dimz) == thread_id);

    //printf("\n i= %d, j= %d, k= %d x= %f, b= %f, bd= %f", i, j, k, 
    //            x[thread_id], b[thread_id], bd[thread_id]);
    printf("\n i= %d, j= %d, k= %d, x= %f, b= %f", i, j, k, x[thread_id], b[thread_id]);

    /*auto result = coefs[1] * b[thread_id];
    if (thread_id > 0) {
        result += coefs[0] * b[thread_id - 1];
    }
    if (thread_id < size - 1) {
        result += coefs[2] * b[thread_id + 1];
    }
    x[thread_id] = result;*/
}


}  // namespace


template <typename ValueType, typename BoundaryType>
void stencil_kernel(std::size_t size, const BoundaryType *bd, const ValueType *b, ValueType *x,
                    std::size_t dimx, std::size_t dimy, std::size_t dimz)
{
    constexpr auto block_size = 512;
    const auto grid_size = (size + block_size - 1) / block_size;
    stencil_kernel_impl<ValueType, BoundaryType><<<grid_size, block_size>>>(size, bd, b, x, dimx, dimy, dimz);
}

template void stencil_kernel<float, float>(std::size_t size, const float *bd, const float *b, float *x, std::size_t, std::size_t, std::size_t);
template void stencil_kernel<double, float>(std::size_t size, const float *bd, const double *b, double *x, std::size_t, std::size_t, std::size_t);
