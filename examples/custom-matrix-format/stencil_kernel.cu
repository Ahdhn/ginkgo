#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2021, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include <cstdlib>

#include <ginkgo/ginkgo.hpp>


namespace {

template <typename T>
__device__ __forceinline__  T pitch(const T i, const T j, const T k, const T c, const T dim_x, const T dim_y, const T dim_z)
{
    return c * dim_x * dim_y * dim_z + k * dim_y * dim_z + j * dim_y + i;
}


// a parallel CUDA kernel that computes the application of a 3 point stencil
template <typename ValueType, typename BoundaryType>
__global__ void stencil_kernel_impl(std::size_t size, BoundaryType *bd,
                                    const ValueType *input, ValueType *output,
                                    std::size_t dimx, std::size_t dimy,
                                    std::size_t dimz, bool init)
{
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id >= size) {
        return;
    }
    std::size_t k = thread_id / (dimy * dimx);
    const std::size_t blah = thread_id - (k * dimx * dimy);
    std::size_t j = blah / dimx;
    std::size_t i = blah % dimx;

    // assert(pitch(i,j,k,dimx, dimy, dimz) == thread_id);

    // printf("\n i= %d, j= %d, k= %d, output= %f, input= %f, bd= %f", i, j, k,
    //       output[thread_id], input[thread_id], bd[thread_id]);

    auto center_pitch = pitch(i, j, k, std::size_t(0), dimx, dimy, dimz);

    if(!init){
        if (k == 0 || k == dimz - 1) {
            bd[center_pitch] = 0;
        } else {
            bd[center_pitch] = 1;
        }
    }

    const ValueType center = input[center_pitch];

    if (bd[center_pitch] == 0) {
        if (!init) {
            output[center_pitch] = 0;
        } else {
            output[center_pitch] = center;
        }

    } else {
        ValueType sum = 0.0;
        int numNeighb = 0;

        if (i > 0) {
            ++numNeighb;
            sum += input[pitch(i - 1, j, k, std::size_t(0), dimx, dimy, dimz)];
        }

        if (j > 0) {
            ++numNeighb;
            sum += input[pitch(i, j - 1, k, std::size_t(0), dimx, dimy, dimz)];
        }

        if (k > 0) {
            ++numNeighb;
            sum += input[pitch(i, j, k - 1, std::size_t(0), dimx, dimy, dimz)];
        }

        if (i < dimx - 1) {
            ++numNeighb;
            sum += input[pitch(i + 1, j, k, std::size_t(0), dimx, dimy, dimz)];
        }

        if (j < dimy - 1) {
            ++numNeighb;
            sum += input[pitch(i, j + 1, k, std::size_t(0), dimx, dimy, dimz)];
        }

        if (k < dimz - 1) {
            ++numNeighb;
            sum += input[pitch(i, j, k + 1, std::size_t(0), dimx, dimy, dimz)];
        }
        const ValueType invh2 = ValueType(1.0);
        output[center_pitch] =
            (-sum + static_cast<ValueType>(numNeighb) * center) * invh2;
    }
}


}  // namespace


template <typename ValueType, typename BoundaryType>
void stencil_kernel(std::size_t size, BoundaryType *bd,
                    const ValueType *input, ValueType *output, std::size_t dimx,
                    std::size_t dimy, std::size_t dimz, bool init)
{
    constexpr auto block_size = 512;
    const auto grid_size = (size + block_size - 1) / block_size;
    stencil_kernel_impl<ValueType, BoundaryType><<<grid_size, block_size>>>(
        size, bd, input, output, dimx, dimy, dimz, init);
}

template void stencil_kernel<float, float>(std::size_t size, float *bd,
                                           const float *input, float *output,
                                           std::size_t dimx, std::size_t dimy,
                                           std::size_t dimz, bool init);
template void stencil_kernel<double, float>(std::size_t size, float *bd,
                                            const double *input, double *output,
                                            std::size_t dimx, std::size_t dimy,
                                            std::size_t dimz, bool init);
