/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2019, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/lower_trs_kernels.hpp"


#include <hip/hip_runtime.h>
#include <hipsparse.h>


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>


#include "core/matrix/dense_kernels.hpp"
#include "core/synthesizer/implementation_selection.hpp"
#include "cuda/base/cusparse_bindings.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The LOWER_TRS solver namespace.
 *
 * @ingroup lower_trs
 */
namespace lower_trs {


#if (defined(CUDA_VERSION) && (CUDA_VERSION >= 9020))
struct cusp_csrsm2_data {
    int algorithm;
    csrsm2Info_t solve_info;
    hipsparseSolvePolicy_t policy;
    hipsparseMatDescr_t factor_descr;
    size_t factor_work_size;
    void *factor_work_vec;
};
cusp_csrsm2_data cusp_csrsm2_data{};

#elif (defined(CUDA_VERSION) && (CUDA_VERSION < 9020))
struct cusp_csrsm_data {
    cusparseSolveAnalysisInfo_t solve_info;
    hipsparseMatDescr_t factor_descr;
};
cusp_csrsm_data cusp_csrsm_data{};
#endif


void clear(std::shared_ptr<const CudaExecutor> exec)
{
#if (defined(CUDA_VERSION) && (CUDA_VERSION >= 9020))
    cusparse::destroy(cusp_csrsm2_data.factor_descr);
    if (cusp_csrsm2_data.solve_info) {
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseDestroyCsrsm2Info(cusp_csrsm2_data.solve_info));
    }
    if (cusp_csrsm2_data.factor_work_vec != nullptr) {
        exec->free(cusp_csrsm2_data.factor_work_vec);
    }
#elif (defined(CUDA_VERSION) && (CUDA_VERSION < 9020))
    cusparse::destroy(cusp_csrsm_data.factor_descr);
    GKO_ASSERT_NO_CUSPARSE_ERRORS(
        cusparseDestroySolveAnalysisInfo(cusp_csrsm_data.solve_info));
#endif
}


template <typename ValueType, typename IndexType>
void generate(std::shared_ptr<const CudaExecutor> exec,
              const matrix::Csr<ValueType, IndexType> *matrix,
              const gko::size_type num_rhs)
{
    if (cusparse::is_supported<ValueType, IndexType>::value) {
        auto handle = exec->get_cusparse_handle();
#if (defined(CUDA_VERSION) && (CUDA_VERSION >= 9020))
        ValueType one = 1.0;
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseCreateCsrsm2Info(&cusp_csrsm2_data.solve_info));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseCreateMatDescr(&cusp_csrsm2_data.factor_descr));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatIndexBase(
            cusp_csrsm2_data.factor_descr, HIPSPARSE_INDEX_BASE_ZERO));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatType(
            cusp_csrsm2_data.factor_descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatDiagType(
            cusp_csrsm2_data.factor_descr, HIPSPARSE_DIAG_TYPE_NON_UNIT));
        cusp_csrsm2_data.algorithm = 0;
        cusp_csrsm2_data.policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        cusparse::buffer_size_ext(
            handle, cusp_csrsm2_data.algorithm,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            matrix->get_size()[0], num_rhs, matrix->get_num_stored_elements(),
            &one, cusp_csrsm2_data.factor_descr, matrix->get_const_values(),
            matrix->get_const_row_ptrs(), matrix->get_const_col_idxs(), nullptr,
            num_rhs, cusp_csrsm2_data.solve_info, cusp_csrsm2_data.policy,
            &cusp_csrsm2_data.factor_work_size);

        // allocate workspace
        cusp_csrsm2_data.factor_work_vec =
            exec->alloc<void *>(cusp_csrsm2_data.factor_work_size);

        cusparse::csrsm2_analysis(
            handle, cusp_csrsm2_data.algorithm,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            matrix->get_size()[0], num_rhs, matrix->get_num_stored_elements(),
            &one, cusp_csrsm2_data.factor_descr, matrix->get_const_values(),
            matrix->get_const_row_ptrs(), matrix->get_const_col_idxs(), nullptr,
            num_rhs, cusp_csrsm2_data.solve_info, cusp_csrsm2_data.policy,
            cusp_csrsm2_data.factor_work_vec);
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));

#elif (defined(CUDA_VERSION) && (CUDA_VERSION < 9020))
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            cusparseCreateSolveAnalysisInfo(&cusp_csrsm_data.solve_info));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseCreateMatDescr(&cusp_csrsm_data.factor_descr));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatIndexBase(
            cusp_csrsm_data.factor_descr, HIPSPARSE_INDEX_BASE_ZERO));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatType(
            cusp_csrsm_data.factor_descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatDiagType(
            cusp_csrsm_data.factor_descr, HIPSPARSE_DIAG_TYPE_NON_UNIT));

        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        cusparse::csrsm_analysis(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix->get_size()[0],
            matrix->get_num_stored_elements(), cusp_csrsm_data.factor_descr,
            matrix->get_const_values(), matrix->get_const_row_ptrs(),
            matrix->get_const_col_idxs(), cusp_csrsm_data.solve_info);
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));
#endif
    } else {
        GKO_NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_LOWER_TRS_GENERATE_KERNEL);


template <typename ValueType, typename IndexType>
void solve(std::shared_ptr<const CudaExecutor> exec,
           const matrix::Csr<ValueType, IndexType> *matrix,
           matrix::Dense<ValueType> *trans_b, matrix::Dense<ValueType> *trans_x,
           const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *x)
{
    using vec = matrix::Dense<ValueType>;
    if (cusparse::is_supported<ValueType, IndexType>::value) {
        ValueType one = 1.0;
        auto handle = exec->get_cusparse_handle();
#if (defined(CUDA_VERSION) && (CUDA_VERSION >= 9020))
        x->copy_from(gko::lend(b));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        cusparse::csrsm2_solve(
            handle, cusp_csrsm2_data.algorithm,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            matrix->get_size()[0], b->get_stride(),
            matrix->get_num_stored_elements(), &one,
            cusp_csrsm2_data.factor_descr, matrix->get_const_values(),
            matrix->get_const_row_ptrs(), matrix->get_const_col_idxs(),
            x->get_values(), b->get_stride(), cusp_csrsm2_data.solve_info,
            cusp_csrsm2_data.policy, cusp_csrsm2_data.factor_work_vec);
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));
#elif (defined(CUDA_VERSION) && (CUDA_VERSION < 9020))
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        if (b->get_stride() == 1) {
            auto temp_b = const_cast<ValueType *>(b->get_const_values());
            cusparse::csrsm_solve(
                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix->get_size()[0],
                b->get_stride(), &one, cusp_csrsm_data.factor_descr,
                matrix->get_const_values(), matrix->get_const_row_ptrs(),
                matrix->get_const_col_idxs(), cusp_csrsm_data.solve_info,
                temp_b, b->get_size()[0], x->get_values(), x->get_size()[0]);
        } else {
            dense::transpose(exec, trans_b, b);
            dense::transpose(exec, trans_x, x);
            cusparse::csrsm_solve(
                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix->get_size()[0],
                trans_b->get_size()[0], &one, cusp_csrsm_data.factor_descr,
                matrix->get_const_values(), matrix->get_const_row_ptrs(),
                matrix->get_const_col_idxs(), cusp_csrsm_data.solve_info,
                trans_b->get_values(), trans_b->get_size()[1],
                trans_x->get_values(), trans_x->get_size()[1]);
            dense::transpose(exec, x, trans_x);
        }
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));

#endif

    } else {
        GKO_NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_LOWER_TRS_SOLVE_KERNEL);


}  // namespace lower_trs
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
