/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2019, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/lower_trs_kernels.hpp"


#include <hip/hip_runtime.h>
#include <hipsparse.h>


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>


#include "core/synthesizer/implementation_selection.hpp"
#include "cuda/base/cusparse_bindings.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The LOWER_TRS solver namespace.
 *
 * @ingroup lower_trs
 */
namespace lower_trs {


#if (defined(CUDA_VERSION) && (CUDA_VERSION > 9100))
struct cusp_csrsm2_data {
    int algorithm;
    csrsm2Info_t solve_info;
    hipsparseSolvePolicy_t policy;
    hipsparseMatDescr_t factor_descr;
    size_t factor_work_size;
    void *factor_work_vec;
};
static cusp_csrsm2_data cusp_csrsm2_data{};

#elif (defined(CUDA_VERSION) && (CUDA_VERSION < 9200))
struct cusp_csrsm_data {
    cusparseSolveAnalysisInfo_t solve_info;
    hipsparseMatDescr_t factor_descr;
};
static cusp_csrsm_data cusp_csrsm_data{};
#endif


template <typename ValueType, typename IndexType>
void generate(std::shared_ptr<const CudaExecutor> exec,
              const matrix::Csr<ValueType, IndexType> *matrix,
              const matrix::Dense<ValueType> *b)
{
    if (cusparse::is_supported<ValueType, IndexType>::value) {
        std::vector<ValueType> one_vec(b->get_stride(), 1.0);
        ValueType one = one_vec[0];
        auto handle = exec->get_cusparse_handle();
#if (defined(CUDA_VERSION) && (CUDA_VERSION > 9100))
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseCreateCsrsm2Info(&cusp_csrsm2_data.solve_info));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseCreateMatDescr(&cusp_csrsm2_data.factor_descr));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatIndexBase(
            cusp_csrsm2_data.factor_descr, HIPSPARSE_INDEX_BASE_ZERO));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatType(
            cusp_csrsm2_data.factor_descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatDiagType(
            cusp_csrsm2_data.factor_descr, HIPSPARSE_DIAG_TYPE_NON_UNIT));
        cusp_csrsm2_data.algorithm = 0;
        cusp_csrsm2_data.policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
#elif (defined(CUDA_VERSION) && (CUDA_VERSION < 9200))

        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            cusparseCreateSolveAnalysisInfo(&cusp_csrsm_data.solve_info));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseCreateMatDescr(&cusp_csrsm_data.factor_descr));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatIndexBase(
            cusp_csrsm_data.factor_descr, HIPSPARSE_INDEX_BASE_ZERO));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatType(
            cusp_csrsm_data.factor_descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
        GKO_ASSERT_NO_CUSPARSE_ERRORS(hipsparseSetMatDiagType(
            cusp_csrsm_data.factor_descr, HIPSPARSE_DIAG_TYPE_NON_UNIT));
#endif

#if (defined(CUDA_VERSION) && (CUDA_VERSION > 9100))
        exec->synchronize();
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        cusparse::buffer_size_ext(
            handle, cusp_csrsm2_data.algorithm,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            matrix->get_size()[0], b->get_stride(),
            matrix->get_num_stored_elements(), &one,
            cusp_csrsm2_data.factor_descr, matrix->get_const_values(),
            matrix->get_const_row_ptrs(), matrix->get_const_col_idxs(),
            b->get_const_values(), b->get_size()[0],
            cusp_csrsm2_data.solve_info, cusp_csrsm2_data.policy,
            &cusp_csrsm2_data.factor_work_size);
        exec->synchronize();

        // allocate workspace
        if (cusp_csrsm2_data.factor_work_vec != nullptr) {
            exec->free(cusp_csrsm2_data.factor_work_vec);
        }
        cusp_csrsm2_data.factor_work_vec =
            exec->alloc<void *>(cusp_csrsm2_data.factor_work_size);

        exec->synchronize();
        cusparse::csrsm2_analysis(
            handle, cusp_csrsm2_data.algorithm,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            matrix->get_size()[0], b->get_stride(),
            matrix->get_num_stored_elements(), &one,
            cusp_csrsm2_data.factor_descr, matrix->get_const_values(),
            matrix->get_const_row_ptrs(), matrix->get_const_col_idxs(),
            b->get_const_values(), b->get_size()[0],
            cusp_csrsm2_data.solve_info, cusp_csrsm2_data.policy,
            cusp_csrsm2_data.factor_work_vec);
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));
        exec->synchronize();
#elif (defined(CUDA_VERSION) && (CUDA_VERSION < 9200))

        exec->synchronize();
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        exec->synchronize();
        cusparse::csrsm_analysis(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix->get_size()[0],
            matrix->get_num_stored_elements(), cusp_csrsm_data.factor_descr,
            matrix->get_const_values(), matrix->get_const_row_ptrs(),
            matrix->get_const_col_idxs(), cusp_csrsm_data.solve_info);
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));
        exec->synchronize();
#endif
    } else {
        GKO_NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_LOWER_TRS_GENERATE_KERNEL);


template <typename ValueType, typename IndexType>
void solve(std::shared_ptr<const CudaExecutor> exec,
           const matrix::Csr<ValueType, IndexType> *matrix,
           const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *x)
{
    if (cusparse::is_supported<ValueType, IndexType>::value) {
        std::vector<ValueType> one_vec(b->get_stride(), 1.0);
        ValueType one = one_vec[0];
        auto handle = exec->get_cusparse_handle();
#if (defined(CUDA_VERSION) && (CUDA_VERSION > 9100))
        exec->copy_from(exec.get(), b->get_size()[0] * b->get_stride(),
                        b->get_const_values(), x->get_values());
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        cusparse::csrsm2_solve(
            handle, cusp_csrsm2_data.algorithm,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
            matrix->get_size()[0], b->get_stride(),
            matrix->get_num_stored_elements(), &one,
            cusp_csrsm2_data.factor_descr, matrix->get_const_values(),
            matrix->get_const_row_ptrs(), matrix->get_const_col_idxs(),
            x->get_values(), b->get_size()[0], cusp_csrsm2_data.solve_info,
            cusp_csrsm2_data.policy, cusp_csrsm2_data.factor_work_vec);
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));
        exec->synchronize();
#elif (defined(CUDA_VERSION) && (CUDA_VERSION < 9200))
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
        cusparse::csrsm_solve(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, matrix->get_size()[0],
            b->get_stride(), &one, cusp_csrsm_data.factor_descr,
            matrix->get_const_values(), matrix->get_const_row_ptrs(),
            matrix->get_const_col_idxs(), cusp_csrsm_data.solve_info,
            b->get_const_values(), b->get_size()[0], x->get_values(),
            x->get_size()[0]);
        GKO_ASSERT_NO_CUSPARSE_ERRORS(
            hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE));
        exec->synchronize();

#endif

    } else {
        GKO_NOT_IMPLEMENTED;
    }
}

GKO_INSTANTIATE_FOR_EACH_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_LOWER_TRS_SOLVE_KERNEL);


}  // namespace lower_trs
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
