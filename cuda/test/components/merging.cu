#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2020, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "cuda/components/merging.cuh"


#include <algorithm>
#include <memory>
#include <random>


#include <gtest/gtest.h>


#include <ginkgo/core/base/array.hpp>
#include <ginkgo/core/base/executor.hpp>


#include "cuda/components/cooperative_groups.cuh"


namespace {


using namespace gko::kernels::cuda;
using namespace cooperative_groups;


class Merging : public ::testing::Test {
protected:
    Merging()
        : ref(gko::ReferenceExecutor::create()),
          cuda(gko::CudaExecutor::create(0, ref)),
          rng(123456),
          rng_runs{100},
          rng_run{},
          max_size{1637},
          sizes{0,  1,  2,   3,   4,   10,  15,   16,
                31, 34, 102, 242, 534, 956, 1239, 1637},
          data1(ref, max_size),
          data2(ref, max_size),
          outdata(ref, 2 * max_size),
          refdata(ref, 2 * max_size),
          ddata1(cuda),
          ddata2(cuda),
          doutdata(cuda, 2 * max_size)
    {}

    void init_data()
    {
        std::uniform_int_distribution<gko::int32> dist(0, max_size);
        std::fill_n(data1.get_data(), max_size, 0);
        std::fill_n(data2.get_data(), max_size, 0);
        for (auto i = 0; i < max_size; ++i) {
            // here we also want to test some corner cases
            // first two runs: zero data1
            if (rng_run > 0) data1.get_data()[i] = dist(rng);
            // first and third run: zero data2
            if (rng_run > 3 || rng_run == 1) data2.get_data()[i] = dist(rng);
        }
        std::sort(data1.get_data(), data1.get_data() + max_size);
        std::sort(data2.get_data(), data2.get_data() + max_size);

        ddata1 = data1;
        ddata2 = data2;
    }

    void assert_eq_ref(int size, int eq_size)
    {
        outdata = doutdata;
        auto out_ptr = outdata.get_const_data();
        auto out_end = out_ptr + eq_size;
        auto ref_ptr = refdata.get_data();
        std::copy_n(data1.get_const_data(), size, ref_ptr);
        std::copy_n(data2.get_const_data(), size, ref_ptr + size);
        std::sort(ref_ptr, ref_ptr + 2 * size);

        ASSERT_TRUE(std::equal(out_ptr, out_end, ref_ptr));
    }

    std::shared_ptr<gko::ReferenceExecutor> ref;
    std::shared_ptr<gko::CudaExecutor> cuda;
    std::default_random_engine rng;

    int rng_runs;
    int rng_run;
    int max_size;
    std::vector<int> sizes;
    gko::Array<gko::int32> data1;
    gko::Array<gko::int32> data2;
    gko::Array<gko::int32> outdata;
    gko::Array<gko::int32> refdata;
    gko::Array<gko::int32> ddata1;
    gko::Array<gko::int32> ddata2;
    gko::Array<gko::int32> doutdata;
};


__global__ void test_merge_step(const gko::int32 *a, const gko::int32 *b,
                                gko::int32 *c)
{
    auto warp = tiled_partition<config::warp_size>(this_thread_block());
    auto i = warp.thread_rank();
    auto result = group_merge_step(a[i], b[i], config::warp_size, warp);
    c[i] = min(result.a_val, result.b_val);
}

TEST_F(Merging, MergeStep)
{
    for (auto i = 0; i < rng_runs; ++i) {
        init_data();
        test_merge_step<<<1, config::warp_size>>>(ddata1.get_const_data(),
                                                  ddata2.get_const_data(),
                                                  doutdata.get_data());

        assert_eq_ref(config::warp_size, config::warp_size);
    }
}


__global__ void test_merge(const gko::int32 *a, const gko::int32 *b, int size,
                           gko::int32 *c)
{
    auto warp = tiled_partition<config::warp_size>(this_thread_block());
    group_merge(a, size, b, size, warp,
                [&](int a_idx, gko::int32 a_val, int b_idx, gko::int32 b_val,
                    int i) { c[i] = min(a_val, b_val); });
}

TEST_F(Merging, FullMerge)
{
    for (auto i = 0; i < rng_runs; ++i) {
        init_data();
        for (auto size : sizes) {
            test_merge<<<1, config::warp_size>>>(ddata1.get_const_data(),
                                                 ddata2.get_const_data(), size,
                                                 doutdata.get_data());

            assert_eq_ref(size, 2 * size);
        }
    }
}


__global__ void test_merge_3way(const gko::int32 *a, const gko::int32 *b,
                                int size, int separator, gko::int32 *c)
{
    auto warp = tiled_partition<config::warp_size>(this_thread_block());
    group_merge_3way(
        a, separator, a + separator, size - separator, b, size, warp,
        [&](int a_idx, gko::int32 a_val, int b_idx, gko::int32 b_val, int i) {
            c[i] = min(a_val, b_val);
        });
}

TEST_F(Merging, FullMerge3Way)
{
    for (auto i = 0; i < rng_runs; ++i) {
        init_data();
        for (auto size : sizes) {
            for (auto separator :
                 {0, 1, size / 3, 2 * size / 3, size - 1, size}) {
                test_merge_3way<<<1, config::warp_size>>>(
                    ddata1.get_const_data(), ddata2.get_const_data(), size,
                    separator, doutdata.get_data());

                assert_eq_ref(size, 2 * size);
            }
        }
    }
}


__global__ void test_sequential_merge(const gko::int32 *a, const gko::int32 *b,
                                      int size, gko::int32 *c)
{
    sequential_merge(
        a, size, b, size,
        [&](int a_idx, gko::int32 a_val, int b_idx, gko::int32 b_val, int i) {
            c[i] = min(a_val, b_val);
        });
}

TEST_F(Merging, SequentialFullMerge)
{
    for (auto i = 0; i < rng_runs; ++i) {
        init_data();
        for (auto size : sizes) {
            test_sequential_merge<<<1, 1>>>(ddata1.get_const_data(),
                                            ddata2.get_const_data(), size,
                                            doutdata.get_data());

            assert_eq_ref(size, 2 * size);
        }
    }
}


__global__ void test_sequential_merge_3way(const gko::int32 *a,
                                           const gko::int32 *b, int size,
                                           int separator, gko::int32 *c)
{
    sequential_merge_3way(
        a, separator, a + separator, size - separator, b, size,
        [&](int a_idx, gko::int32 a_val, int b_idx, gko::int32 b_val, int i) {
            c[i] = min(a_val, b_val);
        });
}

TEST_F(Merging, SequentialFull3WayMerge)
{
    for (auto i = 0; i < rng_runs; ++i) {
        init_data();
        for (auto size : sizes) {
            for (auto separator :
                 {0, 1, size / 3, 2 * size / 3, size - 1, size}) {
                test_sequential_merge_3way<<<1, 1>>>(
                    ddata1.get_const_data(), ddata2.get_const_data(), size,
                    separator, doutdata.get_data());

                assert_eq_ref(size, 2 * size);
            }
        }
    }
}


}  // namespace
