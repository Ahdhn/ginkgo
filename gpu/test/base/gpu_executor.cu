#include <core/base/executor.hpp>


#include <type_traits>


#include <gtest/gtest.h>

#include <core/base/exception_helpers.hpp>

#include <gpu/base/exception.hpp>


#include <gpu/test/base/gpu_kernel.cu>


namespace {


using exec_ptr = std::shared_ptr<gko::Executor>;


TEST(GpuExecutor, AllocatesAndFreesMemory)
{
    const int num_elems = 10;
    auto cpu = gko::CpuExecutor::create();
    auto gpu = gko::GpuExecutor::create(0, cpu);

    ASSERT_EQ(1,cpu->GpuExecutor:getDeviceCount());
    int *ptr ;
    
    ASSERT_NO_THROW(ptr = gpu->alloc<int>(num_elems));
    ASSERT_NO_THROW(gpu->free(ptr));

    ASSERT_NO_THROW(gpu->synchronize()); // allow for synchronization to prevent any errors to be carried forward
}


TEST(GpuExecutor, FailsWhenOverallocating)
{
    const gko::size_type num_elems = 1ll << 50;  // 4PB of integers
    auto cpu = gko::CpuExecutor::create();
    auto gpu = gko::GpuExecutor::create(0, cpu);
    int *ptr ;

    ASSERT_THROW(ptr = gpu->alloc<int>(num_elems), gko::AllocationError);

    gpu->free(ptr);

    ASSERT_NO_THROW(gpu->synchronize());
}


TEST(GpuExecutor, CopiesDataFromCpu)
{
    
    double orig[] = {3,8};
    const int num_elems = std::extent<decltype(orig)>::value;
    auto cpu = gko::CpuExecutor::create();
    auto gpu = gko::GpuExecutor::create(0, cpu);
    double *d_copy = gpu->alloc<double>(num_elems);
   
    double *copy = cpu->alloc<double>(num_elems);
    
    gpu->copy_from(cpu.get(), num_elems, orig, d_copy); //copy data to gpu
    
    run_on_gpu(num_elems, d_copy); //run a gpu kernel
    
    cpu->copy_from(gpu.get(), num_elems, d_copy, copy); //copy back data from gpu
    
    EXPECT_EQ(2.5, copy[0]);
    EXPECT_EQ(5, copy[1]);

    
    ASSERT_NO_THROW(gpu->synchronize());
 
}


}  // namespace
