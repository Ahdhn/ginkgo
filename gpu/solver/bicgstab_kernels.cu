#include "hip/hip_runtime.h"
#include "core/solver/bicgstab_kernels.hpp"


#include "core/base/exception_helpers.hpp"
#include "core/base/math.hpp"
#include "gpu/base/types.hpp"


namespace gko {
namespace kernels {
namespace gpu {
namespace bicgstab {


struct size {
    size_type num_rows_;
    size_type num_cols_;
    constexpr size_type get_num_rows() const noexcept { return num_rows_; }
    constexpr size_type get_num_cols() const noexcept { return num_cols_; }
};


inline int64 ceildiv(int64 a, int64 b) { return (a + b - 1) / b; }


template <typename ValueType>
__global__ void initialize_kernel(size_type m, size_type n, size_type lda,
                                  const ValueType *b, ValueType *r,
                                  ValueType *z, ValueType *p, ValueType *v,
                                  ValueType *t, ValueType *y, ValueType *rr,
                                  ValueType *s, ValueType *prev_rho,
                                  ValueType *rho, ValueType *beta,
                                  ValueType *alpha, ValueType *omega)
{
    const size_type tidx = blockDim.x * blockIdx.x + threadIdx.x;

    if (tidx < n) {
        rho[tidx] = one<ValueType>();
        alpha[tidx] = one<ValueType>();
        beta[tidx] = one<ValueType>();
        omega[tidx] = one<ValueType>();
        prev_rho[tidx] = one<ValueType>();
    }

    if (tidx < m * lda) {
        r[tidx] = b[tidx];
        rr[tidx] = b[tidx];
        z[tidx] = zero<ValueType>();
        p[tidx] = zero<ValueType>();
        v[tidx] = zero<ValueType>();
        t[tidx] = zero<ValueType>();
        s[tidx] = zero<ValueType>();
        y[tidx] = zero<ValueType>();
    }
}


template <typename ValueType>
void initialize(const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *r,
                matrix::Dense<ValueType> *rr, matrix::Dense<ValueType> *y,
                matrix::Dense<ValueType> *s, matrix::Dense<ValueType> *t,
                matrix::Dense<ValueType> *z, matrix::Dense<ValueType> *v,
                matrix::Dense<ValueType> *p, matrix::Dense<ValueType> *prev_rho,
                matrix::Dense<ValueType> *rho, matrix::Dense<ValueType> *alpha,
                matrix::Dense<ValueType> *beta, matrix::Dense<ValueType> *omega)
{
    constexpr int block_size_x = 512;
    const dim3 block_size(block_size_x, 1, 1);
    const dim3 grid_size(
        ceildiv(b->get_num_rows() * b->get_padding(), block_size.x), 1, 1);

    initialize_kernel<<<grid_size, block_size, 0, 0>>>(
        b->get_num_rows(), b->get_num_cols(), b->get_padding(),
        as_cuda_type(b->get_const_values()), as_cuda_type(r->get_values()),
        as_cuda_type(z->get_values()), as_cuda_type(p->get_values()),
        as_cuda_type(v->get_values()), as_cuda_type(t->get_values()),
        as_cuda_type(y->get_values()), as_cuda_type(rr->get_values()),
        as_cuda_type(s->get_values()), as_cuda_type(prev_rho->get_values()),
        as_cuda_type(rho->get_values()), as_cuda_type(beta->get_values()),
        as_cuda_type(alpha->get_values()), as_cuda_type(omega->get_values()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BICGSTAB_INITIALIZE_KERNEL);


template <typename ValueType>
__global__ void step_1_kernel(size_type m, size_type n, size_type lda,
                              ValueType *p, const ValueType *r,
                              const ValueType *v, const ValueType *rho,
                              const ValueType *prev_rho, const ValueType *alpha,
                              const ValueType *omega)
{
    const size_type tidx = blockDim.x * blockIdx.x + threadIdx.x;
    const size_type col = tidx % lda;
    ValueType tmp = zero<ValueType>();


    if (tidx < m * lda) {
        tmp = rho[col] / prev_rho[col] * alpha[col] / omega[col];
        p[tidx] = (tmp == zero<ValueType>())
                      ? r[tidx]
                      : r[tidx] + tmp * (p[tidx] - omega[col] * v[tidx]);
    }
}


template <typename ValueType>
void step_1(const matrix::Dense<ValueType> *r, matrix::Dense<ValueType> *p,
            const matrix::Dense<ValueType> *v,
            const matrix::Dense<ValueType> *rho,
            const matrix::Dense<ValueType> *prev_rho,
            const matrix::Dense<ValueType> *alpha,
            const matrix::Dense<ValueType> *omega)
{
    constexpr int block_size_x = 512;
    const dim3 block_size(block_size_x, 1, 1);
    const dim3 grid_size(
        ceildiv(p->get_num_rows() * p->get_padding(), block_size.x), 1, 1);

    step_1_kernel<<<grid_size, block_size, 0, 0>>>(
        p->get_num_rows(), p->get_num_cols(), p->get_padding(),
        as_cuda_type(p->get_values()), as_cuda_type(r->get_const_values()),
        as_cuda_type(v->get_const_values()),
        as_cuda_type(rho->get_const_values()),
        as_cuda_type(prev_rho->get_const_values()),
        as_cuda_type(alpha->get_const_values()),
        as_cuda_type(omega->get_const_values()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BICGSTAB_STEP_1_KERNEL);


template <typename ValueType>
__global__ void step_2_kernel(size_type m, size_type n, size_type lda,
                              ValueType *s, const ValueType *r,
                              const ValueType *v, ValueType *alpha,
                              const ValueType *beta, const ValueType *rho)
{
    const size_type tidx = blockDim.x * blockIdx.x + threadIdx.x;
    const size_type col = tidx % lda;
    // ValueType tmp = zero<ValueType>();
    if (tidx < n) {
        alpha[n] = rho[n] / beta[n];
    }
    __syncthreads();
    if (tidx < m * lda) {
        alpha[col] = rho[col] / beta[col];
        s[tidx] = (alpha[col] == zero<ValueType>())
                      ? r[tidx]
                      : r[tidx] - alpha[col] * v[tidx];
    }
}


template <typename ValueType>
void step_2(const matrix::Dense<ValueType> *r, matrix::Dense<ValueType> *s,
            const matrix::Dense<ValueType> *v,
            const matrix::Dense<ValueType> *rho,
            matrix::Dense<ValueType> *alpha,
            const matrix::Dense<ValueType> *beta)
{
    constexpr int block_size_x = 512;
    const dim3 block_size(block_size_x, 1, 1);
    const dim3 grid_size(
        ceildiv(s->get_num_rows() * s->get_padding(), block_size.x), 1, 1);

    step_2_kernel<<<grid_size, block_size, 0, 0>>>(
        s->get_num_rows(), s->get_num_cols(), s->get_padding(),
        as_cuda_type(s->get_values()), as_cuda_type(r->get_const_values()),
        as_cuda_type(v->get_const_values()), as_cuda_type(alpha->get_values()),
        as_cuda_type(beta->get_const_values()),
        as_cuda_type(rho->get_const_values()));
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BICGSTAB_STEP_2_KERNEL);


template <typename ValueType>
__global__ void step_3_kernel(size_type m, size_type n, size_type lda,
                              ValueType *x, ValueType *r, const ValueType *y,
                              const ValueType *z, const ValueType *s,
                              const ValueType *t, ValueType *omega,
                              const ValueType *alpha, const ValueType *beta)
{
    const size_type tidx = blockDim.x * blockIdx.x + threadIdx.x;
    const size_type col = tidx % lda;
    // ValueType tmp = zero<ValueType>();
    if (tidx < n) {
        omega[n] = omega[n] / beta[n];
    }
    __syncthreads();
    if (tidx < m * lda) {
        omega[col] = omega[col] / beta[col];
        // x[tidx] = (omega[col] == zero<ValueType>()) ? x[tidx] : x[tidx] +
        // alpha[col] * y[tidx] + omega[col]*z[tidx];
        x[tidx] = x[tidx] + alpha[col] * y[tidx] + omega[col] * z[tidx];
        // r[tidx] = (omega[col] == zero<ValueType>()) ? r[tidx] : r[tidx] - tmp
        // * q[tidx];
        r[tidx] = s[tidx] - omega[col] * t[tidx];
    }
}

template <typename ValueType>
void step_3(matrix::Dense<ValueType> *x, matrix::Dense<ValueType> *r,
            const matrix::Dense<ValueType> *s,
            const matrix::Dense<ValueType> *t,
            const matrix::Dense<ValueType> *y,
            const matrix::Dense<ValueType> *z,
            const matrix::Dense<ValueType> *alpha,
            const matrix::Dense<ValueType> *beta,
            matrix::Dense<ValueType> *omega)
{
    constexpr int block_size_x = 512;
    const dim3 block_size(block_size_x, 1, 1);
    const dim3 grid_size(
        ceildiv(x->get_num_rows() * x->get_padding(), block_size.x), 1, 1);

    step_3_kernel<<<grid_size, block_size, 0, 0>>>(
        x->get_num_rows(), x->get_num_cols(), x->get_padding(),
        as_cuda_type(x->get_values()), as_cuda_type(r->get_values()),
        as_cuda_type(y->get_const_values()),
        as_cuda_type(z->get_const_values()),
        as_cuda_type(s->get_const_values()),
        as_cuda_type(t->get_const_values()), as_cuda_type(omega->get_values()),
        as_cuda_type(alpha->get_const_values()),
        as_cuda_type(beta->get_const_values()));
}
GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_BICGSTAB_STEP_3_KERNEL);


}  // namespace bicgstab
}  // namespace gpu
}  // namespace kernels
}  // namespace gko
