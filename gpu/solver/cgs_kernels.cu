#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright 2017-2018

Karlsruhe Institute of Technology
Universitat Jaume I
University of Tennessee

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice,
   this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
   may be used to endorse or promote products derived from this software
   without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/cgs_kernels.hpp"


#include "core/base/exception_helpers.hpp"
#include "core/base/math.hpp"
#include "gpu/base/types.hpp"


namespace gko {
namespace kernels {
namespace gpu {
namespace cgs {


constexpr int default_block_size = 512;

/*
template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void initialize_kernel(
    size_type num_rows, size_type stride, const ValueType *__restrict__ b,
    ValueType *__restrict__ r, ValueType *__restrict__ z,
    ValueType *__restrict__ p, ValueType *__restrict__ q,
    ValueType *__restrict__ alpha, ValueType *__restrict__ beta,
    ValueType *__restrict__ gamma, ValueType *__restrict__ prev_rho,
    ValueType *__restrict__ rho)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;

    if (tidx < stride) {
        rho[tidx] = zero<ValueType>();
        prev_rho[tidx] = one<ValueType>();
    }

    if (tidx < num_rows * stride) {
        r[tidx] = b[tidx];
        z[tidx] = zero<ValueType>();
        p[tidx] = zero<ValueType>();
        q[tidx] = zero<ValueType>();
    }
}
*/

template <typename ValueType>
void initialize(std::shared_ptr<const DefaultExecutor> exec,
                const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *r,
                matrix::Dense<ValueType> *r_tld, matrix::Dense<ValueType> *p,
                matrix::Dense<ValueType> *q, matrix::Dense<ValueType> *u,
                matrix::Dense<ValueType> *u_hat,
                matrix::Dense<ValueType> *v_hat, matrix::Dense<ValueType> *t,
                matrix::Dense<ValueType> *alpha, matrix::Dense<ValueType> *beta,
                matrix::Dense<ValueType> *gamma,
                matrix::Dense<ValueType> *prev_rho,
                matrix::Dense<ValueType> *rho)
{
    NOT_IMPLEMENTED;
    // this is the code from the solver template
    /*
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(b->get_num_rows() * b->get_stride(), block_size.x), 1, 1);

    initialize_kernel<<<grid_size, block_size, 0, 0>>>(
        b->get_num_rows(), b->get_stride(),
    as_cudaValueType(b->get_const_values()), as_cudaValueType(r->get_values()),
    as_cudaValueType(z->get_values()), as_cudaValueType(p->get_values()),
    as_cudaValueType(q->get_values()), as_cudaValueType(prev_rho->get_values()),
    as_cudaValueType(rho->get_values()));
    */
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CGS_INITIALIZE_KERNEL);

/*
template <typename ValueType>
__global__ __launch_bounds__(default_block_size) void step_2_kernel(
    size_type num_rows, size_type num_cols, size_type stride,
    size_type x_stride, ValueType *__restrict__ x, ValueType *__restrict__ r,
    const ValueType *__restrict__ p, const ValueType *__restrict__ q,
    const ValueType *__restrict__ beta, const ValueType *__restrict__ rho)
{
    const auto tidx =
        static_cast<size_type>(blockDim.x) * blockIdx.x + threadIdx.x;
    const auto row = tidx / stride;
    const auto col = tidx % stride;

    if (col >= num_cols || tidx >= num_rows * num_cols) {
        return;
    }
    if (beta[col] != zero<ValueType>()) {
        const auto tmp = rho[col] / beta[col];
        x[row * x_stride + col] += tmp * p[tidx];
        r[tidx] -= tmp * q[tidx];
    }
}
*/

template <typename ValueType>
void step_1(std::shared_ptr<const DefaultExecutor> exec,
            const matrix::Dense<ValueType> *r, matrix::Dense<ValueType> *u,
            matrix::Dense<ValueType> *p, const matrix::Dense<ValueType> *q,
            matrix::Dense<ValueType> *beta, const matrix::Dense<ValueType> *rho,
            const matrix::Dense<ValueType> *rho_prev)
{
    NOT_IMPLEMENTED;
    // this is the code from the solver template
    /*
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(p->get_num_rows() * p->get_stride(), block_size.x), 1, 1);

    step_2_kernel<<<grid_size, block_size, 0, 0>>>(
        p->get_num_rows(), p->get_num_cols(), p->get_stride(), x->get_stride(),
        as_cudaValueType(x->get_values()), as_cudaValueType(r->get_values()),
        as_cudaValueType(p->get_const_values()),
        as_cudaValueType(q->get_const_values()),
        as_cudaValueType(beta->get_const_values()),
        as_cudaValueType(rho->get_const_values()));
    */
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CGS_STEP_1_KERNEL);


template <typename ValueType>
void step_2(std::shared_ptr<const DefaultExecutor> exec,
            const matrix::Dense<ValueType> *u,
            const matrix::Dense<ValueType> *v_hat, matrix::Dense<ValueType> *q,
            matrix::Dense<ValueType> *t, matrix::Dense<ValueType> *alpha,
            const matrix::Dense<ValueType> *rho,
            const matrix::Dense<ValueType> *gamma)
{
    NOT_IMPLEMENTED;
    // this is the code from the solver template
    /*
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(p->get_num_rows() * p->get_stride(), block_size.x), 1, 1);

    step_2_kernel<<<grid_size, block_size, 0, 0>>>(
        p->get_num_rows(), p->get_num_cols(), p->get_stride(), x->get_stride(),
        as_cudaValueType(x->get_values()), as_cudaValueType(r->get_values()),
        as_cudaValueType(p->get_const_values()),
        as_cudaValueType(q->get_const_values()),
        as_cudaValueType(beta->get_const_values()),
        as_cudaValueType(rho->get_const_values()));
    */
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CGS_STEP_2_KERNEL);


template <typename ValueType>
void step_3(std::shared_ptr<const DefaultExecutor> exec,
            const matrix::Dense<ValueType> *t,
            const matrix::Dense<ValueType> *u_hat, matrix::Dense<ValueType> *r,
            matrix::Dense<ValueType> *x, const matrix::Dense<ValueType> *alpha)
{
    NOT_IMPLEMENTED;
    // this is the code from the solver template
    /*
    const dim3 block_size(default_block_size, 1, 1);
    const dim3 grid_size(
        ceildiv(p->get_num_rows() * p->get_stride(), block_size.x), 1, 1);

    step_2_kernel<<<grid_size, block_size, 0, 0>>>(
        p->get_num_rows(), p->get_num_cols(), p->get_stride(), x->get_stride(),
        as_cudaValueType(x->get_values()), as_cudaValueType(r->get_values()),
        as_cudaValueType(p->get_const_values()),
        as_cudaValueType(q->get_const_values()),
        as_cudaValueType(beta->get_const_values()),
        as_cudaValueType(rho->get_const_values()));
    */
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CGS_STEP_3_KERNEL);


}  // namespace cgs
}  // namespace gpu
}  // namespace kernels
}  // namespace gko
